﻿// cpp_compiler_options_openmp.cpp
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include "common.h"

volatile DWORD dwStart;
volatile int global = 0;

double test1(int num_steps) {
    int i;
    global++;
    double x, pi, sum = 0.0, step;

    step = 1.0 / (double)num_steps;

    for (i = 1; i <= num_steps; i++) {
        x = (i - 0.5) * step;
        sum = sum + 4.0 / (1.0 + x * x);
    }

    pi = step * sum;
    return pi;
}

double test2(int num_steps) {
    int i;
    global++;
    double x, pi, sum = 0.0, step;

    step = 1.0 / (double)num_steps;

#pragma omp parallel for reduction(+:sum) private(x)
    for (i = 1; i <= num_steps; i++) {
        x = (i - 0.5) * step;
        sum = sum + 4.0 / (1.0 + x * x);
    }

    pi = step * sum;
    return pi;
}

int main(int argc, char* argv[]) {
    double   d;
    int n = 1000000000;

    if (argc > 1)
        n = atoi(argv[1]);

    std::chrono::steady_clock::time_point begin;
    begin = StartTimer();
    dwStart = GetTickCount();
    d = test1(n);
    std::cout << "test1(): For " << n << " steps, pi = " << d << ", " << GetDurationInMicroSeconds(begin, StopTimer()) << " microseconds" << std::endl;
    printf_s("For %d steps, pi = %.15f, %d milliseconds\n", n, d, GetTickCount() - dwStart);

    dwStart = GetTickCount();
    begin = StartTimer();
    d = test2(n);
    std::cout << "test2(): For " << n << " steps, pi = " << d << ", " << GetDurationInMicroSeconds(begin, StopTimer()) << " microseconds" << std::endl;
    printf_s("For %d steps, pi = %.15f, %d milliseconds\n", n, d, GetTickCount() - dwStart);
}